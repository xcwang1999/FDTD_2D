
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cmath>
#include <chrono>

using namespace std;

const int grid_row = 200;
const int grid_col = 200;

void initialize_parameters(double *gaz, double *gi2, double *gi3, double *fi2,
                           double *fi3, double *gj2, double *gj3, double *fj2,
                           double *fj3);
void creat_PML(double *gi2, double *gi3, double *gj2, double *gj3,
               double *fi1, double *fi2, double *fi3, double *fj1,
               double *fj2, double *fj3, int npml);
void creat_cylinders(pair<int, int> *centers, int num, double *gaz, double *gbz,
                     double radius, double epsr, double sigma, double delta_t,
                     double epsz);
__global__ void incident_Ez_values(double *ez_inc, double *hx_inc);
__global__ void absorbing_boundary_condition(double *ez_inc, double *boundary_low, double *boundary_high);
__global__ void inject_source(double *ez_inc, double pulse, int t0, int spread, int time_step);
__global__ void calculate_Dz(double *dz, double *hx, double *hy, double *gi3, double *gj3,
                             double *gi2, double *gj2);
__global__ void incident_Dz(double *dz, double *hx_inc, int ia, int ib, int ja, int jb);
__global__ void calculate_Ez(double *ez, double *dz, double *gaz, double *gbz, double *iz);
__global__ void calculate_incident_Hx(double *hx_inc, double *ez_inc);
__global__ void calculate_Hx(double *ez, double *ihx, double *hx, double *fi1, double *fj2, double *fj3);
__global__ void incident_Hx(double *hx, double *ez_inc, int ia, int ib, int ja, int jb);
__global__ void calculate_Hy(double *ez, double *hy, double *ihy, double *fi2, double *fi3, double *fj1);
__global__ void incident_Hy(double *hy, double *ez_inc, int ia, int ib, int ja, int jb);
int main(){

    const int ia = 10;
    const int ib = grid_row - ia - 1;
    const int ja = 10;
    const int jb = grid_col - ja - 1;

    const double delta_x = 0.01;
    const double delta_t = delta_x / 6e8;

    const double epsz = 8.854e-12;

    double *gaz = new double [grid_row*grid_col];
    double *gbz = new double [grid_row*grid_col];

    double *ez = new double [grid_row*grid_col];
    double *dz = new double [grid_row*grid_col];
    double *hx = new double [grid_row*grid_col];
    double *hy = new double [grid_row*grid_col];

    double *iz = new double [grid_row*grid_col];
    double *ihx = new double [grid_row*grid_col];
    double *ihy = new double [grid_row*grid_col];
    double *ez_inc = new double [grid_col];
    double *hx_inc = new double [grid_col];

    // PML parameters
    const int npml = 10;
    double boundary_low[] = {0, 0};
    double boundary_high[] = {0, 0};
    double *gi2 = new double [grid_row];
    double *gi3 = new double [grid_row];
    double *fi1 = new double [grid_row];
    double *fi2 = new double [grid_row];
    double *fi3 = new double [grid_row];
    double *gj2 = new double [grid_col];
    double *gj3 = new double [grid_col];
    double *fj1 = new double [grid_col];
    double *fj2 = new double [grid_col];
    double *fj3 = new double [grid_col];

    // Dielectric area parameter
    const double epsr = 30;
    const double sigma = 0.3;
    const double radius = 10;
    pair<int,int>centers[] = {
        {50, 50}, {50, 100}, {50, 150},
        {100, 50}, {100, 100}, {100, 150},
        {150, 50}, {150, 100}, {150, 150}
    };

    // pulse parameters
    const double t0 = 20;
    const double spread = 8;
    double pulse = 0;

    initialize_parameters((double *)gaz, gi2, gi3, fi2,
                           fi3, gj2, gj3, fj2, fj3);
    creat_PML(gi2, gi3, gj2, gj3, fi1, fi2, fi3, fj1,fj2, fj3, npml);
    creat_cylinders(centers, sizeof(centers)/sizeof(centers[1]), (double *)gaz,
                    (double *)gbz, radius, epsr, sigma, delta_t, epsz);
    double *ez_device, *dz_device, *hx_device, *hy_device, *iz_device,
            *ihx_device, *ihy_device, *ez_inc_device, *hx_inc_device;
    double *gaz_device, *gbz_device;
    double *gi2_device, *gi3_device, *fi1_device, *fi2_device, *fi3_device,
            *gj2_device, *gj3_device, *fj1_device, *fj2_device, *fj3_device;
    double *boundary_low_device, *boundary_high_device;

    hipMalloc((void **)&ez_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&dz_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&hx_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&hy_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&iz_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&ihx_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&ihy_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&ez_inc_device, sizeof(double)*grid_col);
    hipMalloc((void **)&hx_inc_device, sizeof(double)*grid_col);
    hipMalloc((void **)&gaz_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&gbz_device, sizeof(double)*grid_row*grid_col);
    hipMalloc((void **)&gi2_device, sizeof(double)*grid_row);
    hipMalloc((void **)&gi3_device, sizeof(double)*grid_row);
    hipMalloc((void **)&fi1_device, sizeof(double)*grid_row);
    hipMalloc((void **)&fi2_device, sizeof(double)*grid_row);
    hipMalloc((void **)&fi3_device, sizeof(double)*grid_row);
    hipMalloc((void **)&gj2_device, sizeof(double)*grid_col);
    hipMalloc((void **)&gj3_device, sizeof(double)*grid_col);
    hipMalloc((void **)&fj1_device, sizeof(double)*grid_col);
    hipMalloc((void **)&fj2_device, sizeof(double)*grid_col);
    hipMalloc((void **)&fj3_device, sizeof(double)*grid_col);
    hipMalloc((void **)&boundary_low_device, sizeof(double)*2);
    hipMalloc((void **)&boundary_high_device, sizeof(double)*2);

    hipMemcpy(ez_device, ez, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(dz_device, dz, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(hx_device, hx, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(hy_device, hy, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(iz_device, iz, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(ihx_device, ihx, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(ihy_device, ihy, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(ez_inc_device, ez_inc, sizeof(double)*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(hx_inc_device, hx_inc, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(gaz_device, gaz, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(gbz_device, gbz, sizeof(double)*grid_row*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(gi2_device, gi2, sizeof(double)*grid_row, hipMemcpyHostToDevice);
    hipMemcpy(gi3_device, gi3, sizeof(double)*grid_row, hipMemcpyHostToDevice);
    hipMemcpy(fi1_device, fi1, sizeof(double)*grid_row, hipMemcpyHostToDevice);
    hipMemcpy(fi2_device, fi2, sizeof(double)*grid_row, hipMemcpyHostToDevice);
    hipMemcpy(fi3_device, fi3, sizeof(double)*grid_row, hipMemcpyHostToDevice);
    hipMemcpy(gj2_device, gj2, sizeof(double)*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(gj3_device, gj3, sizeof(double)*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(fj1_device, fj1, sizeof(double)*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(fj2_device, fj2, sizeof(double)*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(fj3_device, fj3, sizeof(double)*grid_col, hipMemcpyHostToDevice);
    hipMemcpy(boundary_low_device, boundary_low, sizeof(double)*2, hipMemcpyHostToDevice);
    hipMemcpy(boundary_high_device, boundary_high, sizeof(double)*2, hipMemcpyHostToDevice);

    const int nsteps = 2000;
    dim3 block_size(32, 32);
    dim3 grid_size((grid_col-1)/block_size.x + 1, (grid_row-1)/block_size.y + 1);
    // Main FDTD loop
    for(int time_step=1; time_step<=nsteps; time_step++){
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipEventQuery(start);

        incident_Ez_values<<<(grid_col-1)/128+1, 128>>>(ez_inc_device, hx_inc_device);

        absorbing_boundary_condition<<<1, 1>>>(ez_inc_device, boundary_low_device, boundary_high_device);

        calculate_Dz<<<grid_size, block_size>>>(dz_device, hx_device, hy_device, gi3_device, gj3_device, gi2_device, gj2_device);

        inject_source<<<1, 1>>>(ez_inc_device, pulse, t0, spread, time_step);

        incident_Dz<<<(grid_row-1)/128+1, 128>>>(dz_device, hx_inc_device, ia, ib, ja, jb);

        calculate_Ez<<<grid_size, block_size>>>(ez_device, dz_device, gaz_device, gbz_device, iz_device);

        calculate_incident_Hx<<<(grid_col-1)/128 + 1, 128>>>(hx_inc_device, ez_inc_device);

        calculate_Hx<<<grid_size, block_size>>>(ez_device, ihx_device, hx_device, fi1_device, fj2_device, fj3_device);

        incident_Hx<<<(grid_row-1)/128 + 1, 128>>>(hx_device, ez_inc_device, ia, ib, ja, jb);

        calculate_Hy<<<grid_size, block_size>>>(ez_device, hy_device, ihy_device, fi2_device, fi3_device, fj1_device);

        incident_Hy<<<(grid_col-1)/128 + 1, 128>>>(hy_device, ez_inc_device, ia, ib, ja, jb);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        ofstream outfile;
        outfile.open("execution_time_GPU.txt", ios::app);
        outfile << elapsed_time/1000 << " ";
        outfile.close();

//        if(time_step%250==0){
            double *ez_host = (double *)malloc(sizeof(double)*grid_row*grid_col);
            hipMemcpy(ez_host, ez_device, sizeof(double)*grid_row*grid_col, hipMemcpyDeviceToHost);

            outfile.open("data_cu.txt", ios::app);
            for(int i=0; i<grid_col; i++){
                for(int j=0; j<grid_row; j++){
                    if(j!=grid_row-1){
                        outfile << setprecision(4) << fixed << ez_host[i*grid_col+j] << " ";
                    } else{
                        outfile << setprecision(4) << fixed << ez_host[i*grid_col+j] << endl;
                    }
                }
            }
            outfile.close();
            free(ez_host);
//        }
    }
    hipFree(ez_device);
    hipFree(dz_device);
    hipFree(hx_device);
    hipFree(hy_device);
    hipFree(iz_device);
    hipFree(ihx_device);
    hipFree(ihy_device);
    hipFree(ez_inc_device);
    hipFree(hx_inc_device);
    hipFree(gaz_device);
    hipFree(gbz_device);
    hipFree(gi2_device);
    hipFree(gi3_device);
    hipFree(fi1_device);
    hipFree(fi2_device);
    hipFree(fi3_device);
    hipFree(gj2_device);
    hipFree(gj3_device);
    hipFree(fj1_device);
    hipFree(fj2_device);
    hipFree(fj3_device);
    hipFree(boundary_low_device);
    hipFree(boundary_high_device);

    delete[] gaz;
    delete[] gbz;
    delete[] ez;
    delete[] dz;
    delete[] hx;
    delete[] hy;
    delete[] iz;
    delete[] ihx;
    delete[] ihy;
    delete[] ez_inc;
    delete[] hx_inc;
    delete[] gi2;
    delete[] gi3;
    delete[] fi1;
    delete[] fi2;
    delete[] fi3;
    delete[] gj2;
    delete[] gj3;
    delete[] fj1;
    delete[] fj2;
    delete[] fj3;
    return 0;
}
void initialize_parameters(double *gaz, double *gi2, double *gi3, double *fi2,
                           double *fi3, double *gj2, double *gj3, double *fj2,
                           double *fj3){
    for(int i=0; i<grid_row; i++){
        for(int j=0; j<grid_col; j++){
            gaz[i*grid_col+j] = 1;
        }
    }
    for(int n=0; n<grid_row; n++){
        gi2[n] = 1;
        gi3[n] = 1;
        fi2[n] = 1;
        fi3[n] = 1;
    }
    for(int n=0; n<grid_col; n++){
        gj2[n] = 1;
        gj3[n] = 1;
        fj2[n] = 1;
        fj3[n] = 1;
    }
}

void creat_PML(double *gi2, double *gi3, double *gj2, double *gj3,
               double *fi1, double *fi2, double *fi3, double *fj1,
               double *fj2, double *fj3, int npml){
    for(int n=0; n<npml; n++){
        double xnum = npml - n;
        double xxn = xnum / npml;
        double xn = 0.33 * pow(xxn, 3);
        gi2[n] = 1 / (1 + xn);
        gi2[grid_row-1-n] = 1 / (1 + xn);
        gi3[n] = (1 - xn) / (1 + xn);
        gi3[grid_row-1-n] = (1 - xn) / (1 + xn);
        gj2[n] = 1 / (1 + xn);
        gj2[grid_col-1-n] = 1 / (1 + xn);
        gj3[n] = (1 - xn) / (1 + xn);
        gj3[grid_col-1-n] = (1 - xn) / (1 + xn);
        xxn = (xnum - 0.5) / npml;
        xn = 0.33 * pow(xxn, 3);
        fi1[n] = xn;
        fi1[grid_row-2-n] = xn;
        fi2[n] = 1 / (1 + xn);
        fi2[grid_row-2-n] = 1 / (1 + xn);
        fi3[n] = (1 - xn) / (1 + xn);
        fi3[grid_row-2-n] = (1 - xn) / (1 + xn);
        fj1[n] = xn;
        fj1[grid_col-2-n] = xn;
        fj2[n] = 1 / (1 + xn);
        fj2[grid_col-2-n] = 1 / (1 + xn);
        fj3[n] = (1 - xn) / (1 + xn);
        fj3[grid_col-2-n] = (1 - xn) / (1 + xn);
    }
}

void creat_cylinders(pair<int, int> *centers, int num, double *gaz, double *gbz,
                     double radius, double epsr, double sigma, double delta_t,
                     double epsz){
    for(int n=0; n<num; n++){
        int x = centers[n].first;
        int y = centers[n].second;
        for(int i=0;i<grid_row;i++){
            for(int j=0; j<grid_col; j++){
                double dist = sqrt(pow((x-i),2) + pow((y-j),2));
                if (dist <= radius) {
                    gaz[i*grid_col+j] = 1 / (epsr + (sigma * delta_t / epsz));
                    gbz[i*grid_col+j] = (sigma * delta_t / epsz);
                }
            }
        }
    }
}

__global__ void incident_Ez_values(double *ez_inc, double *hx_inc){
    int j = 1 + blockIdx.x*blockDim.x + threadIdx.x;
    if(j < grid_col){
        ez_inc[j] = ez_inc[j] + 0.5 * (hx_inc[j-1] - hx_inc[j]);
    }
}

__global__ void absorbing_boundary_condition(double *ez_inc, double *boundary_low, double *boundary_high){
    ez_inc[0] = boundary_low[0];
    boundary_low[0]=boundary_low[1];
    boundary_low[1]=ez_inc[1];

    ez_inc[grid_col-1] = boundary_high[0];
    boundary_high[0]=boundary_high[1];
    boundary_high[1]=ez_inc[grid_col-2];
}

__global__ void calculate_Dz(double *dz, double *hx, double *hy, double *gi3, double *gj3,
                             double *gi2, double *gj2){
    int i = 1 + blockIdx.x*blockDim.x + threadIdx.x;
    int j = 1 + blockIdx.y*blockDim.y + threadIdx.y;
    if(i<grid_row && j<grid_col){
        dz[i*grid_col+j] = gi3[i] * gj3[j] * dz[i*grid_col+j] + gi2[i] * gj2[j] * 0.5 *
                    (hy[i*grid_col+j] - hy[(i-1)*grid_col+j] - hx[i*grid_col+j] + hx[i*grid_col+j-1]);

    }
}

__global__ void inject_source(double *ez_inc, double pulse, int t0, int spread, int time_step){
    pulse = exp(-0.5 * pow(((t0-time_step) / spread), 2));
    ez_inc[3] = pulse;
}

__global__ void incident_Dz(double *dz, double *hx_inc, int ia, int ib, int ja, int jb){
    int i = ia + blockIdx.x*blockDim.x + threadIdx.x;
    if(i <= ib){
        dz[i*grid_col+ja] = dz[i*grid_col+ja] + 0.5 * hx_inc[ja-1];
        dz[i*grid_col+jb] = dz[i*grid_col+jb] - 0.5 * hx_inc[jb];
    }
}

__global__ void calculate_Ez(double *ez, double *dz, double *gaz, double *gbz, double *iz){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i<grid_row && j<grid_col){
        ez[i*grid_col+j] = gaz[i*grid_col+j] * (dz[i*grid_col+j] - iz[i*grid_col+j]);
        iz[i*grid_col+j] = iz[i*grid_col+j] + gbz[i*grid_col+j] * ez[i*grid_col+j];
    }
}

__global__ void calculate_incident_Hx(double *hx_inc, double *ez_inc){
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    if(j<grid_col-1){
        hx_inc[j] = hx_inc[j] + 0.5 * (ez_inc[j] - ez_inc[j+1]);
    }
}

__global__ void calculate_Hx(double *ez, double *ihx, double *hx, double *fi1, double *fj2, double *fj3){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i<grid_row && j<(grid_col-1)){
        double curl_e = ez[i*grid_col+j] - ez[i*grid_col+j+1];
        ihx[i*grid_col+j] = ihx[i*grid_col+j] + curl_e;
        hx[i*grid_col+j] = fj3[j] * hx[i*grid_col+j] + fj2[j] * (0.5 * curl_e + fi1[i] * ihx[i*grid_col+j]);

    }
}

__global__ void incident_Hx(double *hx, double *ez_inc, int ia, int ib, int ja, int jb){
    int i = ia + blockIdx.x*blockDim.x + threadIdx.x;
    if(i <= ib){
        hx[i*grid_col+ja-1] = hx[i*grid_col+ja-1] + 0.5 * ez_inc[ja];
        hx[i*grid_col+jb] = hx[i*grid_col+jb] - 0.5 * ez_inc[jb];
    }

}

__global__ void calculate_Hy(double *ez, double *hy, double *ihy, double *fi2, double *fi3, double *fj1){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i<grid_row-1 &&  j<grid_col){
        double curl_e = ez[i*grid_col+j] - ez[(i+1)*grid_col+j];
        ihy[i*grid_col+j] = ihy[i*grid_col+j] + curl_e;
        hy[i*grid_col+j] = fi3[i] * hy[i*grid_col+j] - fi2[i] * (0.5 * curl_e + fj1[j] * ihy[i*grid_col+j]);

    }
}

__global__ void incident_Hy(double *hy, double *ez_inc, int ia, int ib, int ja, int jb){
    int j = ja + blockIdx.x*blockDim.x + threadIdx.x;
    if(j <= jb){
        hy[(ia-1)*grid_col+j] = hy[(ia-1)*grid_col+j] - 0.5 * ez_inc[j];
        hy[ib*grid_col+j] = hy[ib*grid_col+j] + 0.5 * ez_inc[j];
    }
}
